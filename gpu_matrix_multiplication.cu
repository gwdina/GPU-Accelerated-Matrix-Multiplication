// gpu_matrix_multiplication.cu
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define N 1024  // Define size of matrix (N x N)

__global__ void matrixMultiply(float* A, float* B, float* C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        float sum = 0.0f;
        for (int k = 0; k < width; ++k) {
            sum += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = sum;
    }
}

void randomInit(float* data, int size) {
    for (int i = 0; i < size; ++i) {
        data[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main() {
    size_t size = N * N * sizeof(float);

    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    randomInit(h_A, N * N);
    randomInit(h_B, N * N);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x,
                 (N + dimBlock.y - 1) / dimBlock.y);

    matrixMultiply<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    std::cout << "Result (C[0][0]): " << h_C[1] << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
